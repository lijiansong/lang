// NV GPU compute capability detection.
// build: nvcc arch-test.cu && ./a.out
//
// Refs: <https://github.com/BVLC/caffe/blob/master/cmake/Cuda.cmake#L18-L31>

#include <hip/hip_runtime.h>
#include <cstdio>
int main()
{
  int count = 0;
  if (hipSuccess != hipGetDeviceCount(&count)) return -1;
  if (count == 0) return -1;
  for (int device = 0; device < count; ++device)
  {
    hipDeviceProp_t prop;
    if (hipSuccess == hipGetDeviceProperties(&prop, device))
      std::printf("%d.%d \n", prop.major, prop.minor);
  }
  return 0;
}
